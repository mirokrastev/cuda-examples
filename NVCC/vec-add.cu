
#include <hip/hip_runtime.h>
#include <iostream>
#define N 65536

using namespace std;

__global__ void vectorAddition(int* a, int* b, int* c) {
    int id =  blockDim.x * blockIdx.x + threadIdx.x;
    if(N >= id) {
        c[id] = a[id] + b[id];
    }
}

int main() {
    const size_t vectorSize = N * sizeof(int);

    // Allocate memory for host vectors <> Vectors in main memory
    int* hostVectorA = (int*)malloc(vectorSize);
    int* hostVectorB = (int*)malloc(vectorSize);
    int* hostVectorC = (int*)malloc(vectorSize);

    // Allocate memory for device vectors <> Vectors in the GPU memory
    int *deviceVectorA, *deviceVectorB, *deviceVectorC;
    hipMalloc(&deviceVectorA, vectorSize);
    hipMalloc(&deviceVectorB, vectorSize);
    hipMalloc(&deviceVectorC, vectorSize);

    for (int i = 0; i < N; i++) {
        hostVectorA[i] = i;
        hostVectorB[i] = i + 10;
    }

    // Copy data from CPU Array to GPU Array
    hipMemcpy(deviceVectorA, hostVectorA, vectorSize, hipMemcpyHostToDevice);
    hipMemcpy(deviceVectorB, hostVectorB, vectorSize, hipMemcpyHostToDevice);

    // Execution parameters
    const int threadsPerBlock = 256;
    const int blocksPerGrid = N / threadsPerBlock;

    // Launch kernel
    vectorAddition<<<blocksPerGrid, threadsPerBlock>>>(deviceVectorA, deviceVectorB, deviceVectorC);

    // Copy data from GPU Array to CPU Array
    hipMemcpy(hostVectorC, deviceVectorC, vectorSize, hipMemcpyDeviceToHost);

    // Free CPU Memory
    free(hostVectorA);
    free(hostVectorB);
    free(hostVectorC);

    // Free GPU Memory
    hipFree(deviceVectorA);
    hipFree(deviceVectorB);
    hipFree(deviceVectorC);
}
