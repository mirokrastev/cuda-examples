
#include <hip/hip_runtime.h>
# include <iostream>

using namespace std;

__global__ void helloFromGPU() {
    printf("Hello from the GPU!\n");
}

int main() {
    cout << "Hello from the CPU!" << endl;
    helloFromGPU<<<1, 1>>>();
}

/*
<<< Hello from the CPU!
<<< Hello from the GPU!
*/
